#include "hip/hip_runtime.h"
#include "kernel.h"


////////////////////////////////
__global__ void calc_norms(Point *dev_points, const Centroid *const dev_centroids, const int *const dev_centroids_size)
{
	int block_size = blockDim.x;

	double temp_norm, pointX, pointY, centroidX, centroidY;

	if(!dev_points[blockIdx.x*block_size + threadIdx.x].isDummyPoint)
	{
		pointX = dev_points[blockIdx.x*block_size + threadIdx.x].x;
		pointY = dev_points[blockIdx.x*block_size + threadIdx.x].y;

		//init first norm
		centroidX = dev_centroids[0].x;
		centroidY = dev_centroids[0].y;
		dev_points[blockIdx.x*block_size + threadIdx.x].min_dis = sqrt( pow((pointX-centroidX),2) + pow((pointY-centroidY),2) );
		dev_points[blockIdx.x*block_size + threadIdx.x].centroid_index = 0;

		//do the rest
		for(int i=1; i<(*dev_centroids_size); i++)
		{
			centroidX = dev_centroids[i].x;
			centroidY = dev_centroids[i].y;

			temp_norm =  sqrt( pow(pointX-centroidX,2) + pow(pointY-centroidY,2) );

			if(temp_norm < dev_points[blockIdx.x*block_size + threadIdx.x].min_dis)
			{
				dev_points[blockIdx.x*block_size + threadIdx.x].min_dis = temp_norm;
				dev_points[blockIdx.x*block_size + threadIdx.x].centroid_index = i;
			}	
		}
	}
}

__global__ void do_reduction(double *dev_reduction_arr)
{
	int nTotalThreads = blockDim.x;
	int halfPoint;
	int j;
	double temp_res;

	while (nTotalThreads > 1) 
	{
		halfPoint = nTotalThreads >> 1;
		if (threadIdx.x < halfPoint) 
		{
			j = threadIdx.x + halfPoint;
			temp_res = dev_reduction_arr[j];
			if (temp_res < dev_reduction_arr[threadIdx.x]) 
			{
				dev_reduction_arr[threadIdx.x] = temp_res;
			}
		}

		__syncthreads();

		nTotalThreads = nTotalThreads >> 1;

	}//while
}
////////////////////////////////


////////////////////////////////
void check_cuda_status(const hipError_t *const cudaStatus, const char *const error_msg, const int line_number)
{
	if (*cudaStatus != hipSuccess) 
	{
		printf("%d-%s\n", (line_number-1), error_msg);
		printf("%s\n", hipGetErrorString(*cudaStatus));
		system("PAUSE");
		exit(1);
	}
}

void alloc_params_for_cuda(
	Point **dev_points_ptr, const int points_size, 
	Centroid **dev_centroids_ptr, const int centroids_size,
	int ** dev_centroids_size_ptr)
{
	hipError_t cudaStatus;

	cudaStatus = hipMalloc((void**)dev_points_ptr, sizeof(Point)*points_size);
	check_cuda_status(&cudaStatus, "hipMalloc failed! :: alloc_params_for_cuda", __LINE__);

	cudaStatus = hipMalloc((void**)dev_centroids_ptr, sizeof(Centroid)*centroids_size);
	check_cuda_status(&cudaStatus, "hipMalloc failed! :: alloc_params_for_cuda", __LINE__);

	cudaStatus = hipMalloc((void**)dev_centroids_size_ptr, sizeof(int));
	check_cuda_status(&cudaStatus, "hipMalloc failed! :: alloc_params_for_cuda", __LINE__);

	//one immediate initialization
	cudaStatus = hipMemcpy((*dev_centroids_size_ptr), &centroids_size, sizeof(int), hipMemcpyHostToDevice);
	check_cuda_status(&cudaStatus, "hipMemcpy failed! :: alloc_params_for_cuda", __LINE__);
}

void free_params_for_cuda(Point *dev_points, Centroid *dev_centroids, int* dev_centroids_size)
{
	hipFree(dev_points);
	hipFree(dev_centroids);
	hipFree(dev_centroids_size);
}

int get_max_num_of_threads_per_block()
{
	struct hipDeviceProp_t properties;
	hipGetDeviceProperties(&properties, 0);
	return properties.maxThreadsPerBlock;
}
////////////////////////////////


////////////////////////////////
void calc_points_norms(
	Point *points, Centroid *centroids, 
	Point * dev_points, Centroid *dev_centroids, 
	const int points_size, const int centroids_size, 
	int *dev_centroids_size)
{
	hipError_t cudaStatus;

	int max_num_of_thr_per_block = get_max_num_of_threads_per_block();
	int num_of_blocks = points_size/max_num_of_thr_per_block;

	dim3 BlockDim(max_num_of_thr_per_block, 1, 1);
	dim3 GridDim(num_of_blocks, 1, 1);


	cudaStatus = hipMemcpy(dev_points, points, points_size * sizeof(Point), hipMemcpyHostToDevice);
	check_cuda_status(&cudaStatus, "hipMemcpy failed! :: init_params_for_cuda", __LINE__);

	cudaStatus = hipMemcpy(dev_centroids, centroids, centroids_size * sizeof(Centroid), hipMemcpyHostToDevice);
	check_cuda_status(&cudaStatus, "hipMemcpy failed! :: init_params_for_cuda", __LINE__);

	//kernel launch
	calc_norms<<<GridDim, BlockDim>>>(dev_points, dev_centroids, dev_centroids_size);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	check_cuda_status(&cudaStatus, "hipGetLastError failed! :: calc_points_norms", __LINE__);

	// hipDeviceSynchronize waits for the kernel to finish, and returns any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	check_cuda_status(&cudaStatus, "hipDeviceSynchronize failed! :: calc_points_norms", __LINE__);

	cudaStatus = hipMemcpy(points, dev_points, points_size * sizeof(Point), hipMemcpyDeviceToHost);
	check_cuda_status(&cudaStatus, "hipMemcpy failed! :: init_params_for_cuda", __LINE__);	
}

double calc_min_dis_by_reduction(double *const min_distances_arr, const int min_distances_arr_size)
{
	hipError_t cudaStatus;
	double *dev_reduction_arr = NULL;
	double result = -1;
	int reduction_arr_size = min_distances_arr_size;

	find_nearest_power_of_2(&reduction_arr_size);

	double *reduction_arr = (double*)malloc(sizeof(double)*reduction_arr_size);
	memcpy(reduction_arr, min_distances_arr, min_distances_arr_size * sizeof(double));

	for (int i = min_distances_arr_size; i < reduction_arr_size; i++)
	{
		reduction_arr[i] = min_distances_arr[i - min_distances_arr_size];
	}

	cudaStatus = hipMalloc((void**)&dev_reduction_arr, reduction_arr_size * sizeof(double));
	check_cuda_status(&cudaStatus, "hipMalloc failed! :: calc_min_dis_by_reduction", __LINE__);

	cudaStatus = hipMemcpy(dev_reduction_arr, reduction_arr, reduction_arr_size * sizeof(double), hipMemcpyHostToDevice);
	check_cuda_status(&cudaStatus, "hipMemcpy failed! :: calc_min_dis_by_reduction", __LINE__);

	//kernel call
	do_reduction<<<1, reduction_arr_size>>>(dev_reduction_arr);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	check_cuda_status(&cudaStatus, "hipGetLastError failed! :: calc_min_dis_by_reduction", __LINE__);

	// hipDeviceSynchronize waits for the kernel to finish, and returns any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	check_cuda_status(&cudaStatus, "hipDeviceSynchronize failed! :: calc_min_dis_by_reduction", __LINE__);

	cudaStatus = hipMemcpy(reduction_arr, dev_reduction_arr, reduction_arr_size * sizeof(double), hipMemcpyDeviceToHost);
	check_cuda_status(&cudaStatus, "hipMemcpy failed! :: calc_min_dis_by_reduction", __LINE__);

	result = reduction_arr[0] ;

	hipFree(dev_reduction_arr);
	free(reduction_arr);

	return result;
}
////////////////////////////////




