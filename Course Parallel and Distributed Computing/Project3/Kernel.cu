#include "hip/hip_runtime.h"
#include "Kernel.h"


////////////////////////////////
__global__ void calc_norms(Point * dev_points, const int points_size, Centroid* dev_centroids, const int centroids_size) {
	int block_size = blockDim.x;

	float temp_norm, pointX, pointY, centroidX, centroidY;

	if (!dev_points[blockIdx.x*block_size + threadIdx.x].isDummyPoint)
	{
		pointX = dev_points[blockIdx.x*block_size + threadIdx.x].x;
		pointY = dev_points[blockIdx.x*block_size + threadIdx.x].y;

		//init first norm
		centroidX = dev_centroids[0].x;
		centroidY = dev_centroids[0].y;
		dev_points[blockIdx.x*block_size + threadIdx.x].minDisFromCentroid = sqrt(pow((pointX - centroidX), 2) + pow((pointY - centroidY), 2));
		dev_points[blockIdx.x*block_size + threadIdx.x].myCentroidIndex = 0;

		//do the rest
		for (int i = 1; i<centroids_size; i++)
		{
			centroidX = dev_centroids[i].x;
			centroidY = dev_centroids[i].y;

			temp_norm = sqrt(pow(pointX - centroidX, 2) + pow(pointY - centroidY, 2));

			if (temp_norm < dev_points[blockIdx.x*block_size + threadIdx.x].minDisFromCentroid)
			{
				dev_points[blockIdx.x*block_size + threadIdx.x].minDisFromCentroid = temp_norm;
				dev_points[blockIdx.x*block_size + threadIdx.x].myCentroidIndex = i;
			}
		}
	}
}
////////////////////////////////

////////////////////////////////
void check_cuda_status(const hipError_t *const cudaStatus, const char *const error_msg, const int line_number)
{
	if (*cudaStatus != hipSuccess)
	{
		printf("%d-%s\n", (line_number - 1), error_msg);
		printf("%s\n", hipGetErrorString(*cudaStatus));
		system("PAUSE");
		exit(1);
	}
}

int get_max_num_of_threads_per_block()
{
	struct hipDeviceProp_t properties;
	hipGetDeviceProperties(&properties, 0);
	return properties.maxThreadsPerBlock;
}

void set_points_to_work_with_cuda(Point **points_ptr, int *points_size_ptr)
{
	int current_points_size = *points_size_ptr;
	int max_thr_per_block = get_max_num_of_threads_per_block();
	int left_points = -1;

	//in case: we dont have an integer number of blocks
	if ((left_points = (*points_size_ptr) % max_thr_per_block) != 0)
	{
		*points_size_ptr = current_points_size + (max_thr_per_block - left_points);
		(*points_ptr) = (Point*)realloc((*points_ptr), sizeof(Point)*(*points_size_ptr));// shallow copy is ok here
	}
}

void classify_points_by_norm_alloc(Point** dev_points_ptr, const int points_size, Centroid** dev_centroids_ptr, const int centroids_size) {
	static hipError_t cudaStatus;

	cudaStatus = hipMalloc((void**)dev_points_ptr, sizeof(Point)*points_size);
	check_cuda_status(&cudaStatus, "hipMalloc failed! :: classify_points_by_norm_alloc", __LINE__);

	cudaStatus = hipMalloc((void**)dev_centroids_ptr, sizeof(Centroid)*centroids_size);
	check_cuda_status(&cudaStatus, "hipMalloc failed! :: classify_points_by_norm_alloc", __LINE__);
}

void classify_points_by_norm_memCopy(Point* points, const int points_size, Centroid* centroids, const int centroids_size, Point* dev_points, Centroid* dev_centroids) {
	static hipError_t cudaStatus;

	cudaStatus = hipMemcpy(dev_points, points, points_size * sizeof(Point), hipMemcpyHostToDevice);
	check_cuda_status(&cudaStatus, "hipMemcpy failed! :: classify_points_by_norm_memCopy", __LINE__);

	cudaStatus = hipMemcpy(dev_centroids, centroids, centroids_size * sizeof(Centroid), hipMemcpyHostToDevice);
	check_cuda_status(&cudaStatus, "hipMemcpy failed! :: classify_points_by_norm_memCopy", __LINE__);
}

void classify_points_by_norm_free(Point* dev_points, Centroid* dev_centroids) {
	hipFree(dev_points);
	hipFree(dev_centroids);
}

////////////////////////////////

////////////////////////////////
void classify_points_by_norm(Point* points, const int points_size, Centroid* centroids, const int centroids_size) {

	static hipError_t cudaStatus;
	static int max_num_of_thr_per_block = get_max_num_of_threads_per_block();
	static int num_of_blocks = points_size / max_num_of_thr_per_block;// we changed points size to be eqaully divided into max number of blocks for this GPU

	Point *dev_points = NULL;
	Centroid *dev_centroids = NULL;

	classify_points_by_norm_alloc(&dev_points, points_size, &dev_centroids, centroids_size);

	classify_points_by_norm_memCopy(points, points_size, centroids, centroids_size, dev_points, dev_centroids);

	//kernel launch
	calc_norms <<<num_of_blocks, max_num_of_thr_per_block >>> (dev_points, points_size, dev_centroids, centroids_size);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	check_cuda_status(&cudaStatus, "hipGetLastError failed! :: classify_points_by_norm", __LINE__);

	// hipDeviceSynchronize waits for the kernel to finish, and returns any errors encountered during the launch
	cudaStatus = hipDeviceSynchronize();
	check_cuda_status(&cudaStatus, "hipDeviceSynchronize failed! :: classify_points_by_norm", __LINE__);

	cudaStatus = hipMemcpy(points, dev_points, points_size * sizeof(Point), hipMemcpyDeviceToHost);
	check_cuda_status(&cudaStatus, "hipMemcpy failed! :: classify_points_by_norm", __LINE__);

	classify_points_by_norm_free(dev_points, dev_centroids);
}
////////////////////////////////











