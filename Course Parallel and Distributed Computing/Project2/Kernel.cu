#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#include "Kernel.h"


////////////////////////////////
__global__ void calc_norms(
	Point *dev_points,
	const Centroid *const dev_centroids,
	const int centroids_size,
	const int vector_size)
{
	int block_size = blockDim.x;
	double temp_norm;

	if (!dev_points[blockIdx.x*block_size + threadIdx.x].isDummyPoint)
	{
		Point* point = &dev_points[blockIdx.x*block_size + threadIdx.x];

		//init first norm
		temp_norm = 0;
		for (int i = 0; i<vector_size; i++) {
			temp_norm += pow((point->vector[i]) - (dev_centroids[0].vector[i]), 2);
		}
		temp_norm = sqrt(temp_norm);
		point->minDis = temp_norm;
		point->myCentroidIndex = 0;

		//do the rest
		for (int i = 1; i<centroids_size; i++)
		{
			temp_norm = 0;
			for (int j = 0; j<vector_size; j++) {
				temp_norm += pow((point->vector[j]) - (dev_centroids[i].vector[j]), 2);
			}
			temp_norm = sqrt(temp_norm);

			if (temp_norm < (point->minDis))
			{
				point->minDis = temp_norm;
				point->myCentroidIndex = i;
			}
		}
	}
}
////////////////////////////////

////////////////////////////////
void check_cuda_status(const hipError_t *const cudaStatus, const char *const error_msg, const int line_number)
{
	if (*cudaStatus != hipSuccess)
	{
		printf("%d-%s\n", (line_number - 1), error_msg);
		printf("%s\n", hipGetErrorString(*cudaStatus));
		system("PAUSE");
		exit(1);
	}
}

int get_max_num_of_threads_per_block()
{
	struct hipDeviceProp_t properties;
	hipGetDeviceProperties(&properties, 0);
	return properties.maxThreadsPerBlock;
}

/*
* adds dummy-points that will not be taken into considoration
* when doing calculations.
* the dummy-points are added to match the max number of threads and blocks when a kernel lunch is made.
*/
void set_points_to_work_with_cuda(Point **points_ptr, int *points_size_ptr)
{
	int current_points_size = *points_size_ptr;
	int max_thr_per_block = get_max_num_of_threads_per_block();
	int left_points = -1;

	//in case: we dont have an integer number of blocks
	if ((left_points = (*points_size_ptr) % max_thr_per_block) != 0)
	{
		*points_size_ptr = current_points_size + (max_thr_per_block - left_points);
		(*points_ptr) = (Point*)realloc((*points_ptr), sizeof(Point)*(*points_size_ptr));


#pragma omp parallel for default(shared)
		for (int i = current_points_size; i<(*points_size_ptr); i++)
		{
			(*points_ptr)[i].minDis = -1;
			// (*points_ptr)[i].myCentroidIndex = -1;
			(*points_ptr)[i].myCentroidIndex = 101;// set immpossible cluster index for this project def
			(*points_ptr)[i].isDummyPoint = 1;
		}
	}
}

void classify_points_by_norm_alloc(
	Point** dev_points_ptr, const int num_of_points,
	Centroid** dev_centroids_ptr, const int num_of_centroids) {

	static hipError_t cudaStatus;

	cudaStatus = hipMalloc((void**)dev_points_ptr, sizeof(Point)*num_of_points);
	check_cuda_status(&cudaStatus, "hipMalloc failed! :: classify_points_by_norm_alloc", __LINE__);

	cudaStatus = hipMalloc((void**)dev_centroids_ptr, sizeof(Centroid)*num_of_centroids);
	check_cuda_status(&cudaStatus, "hipMalloc failed! :: classify_points_by_norm_alloc", __LINE__);

}

void classify_points_by_norm_memCopy(
	Point* points, const int num_of_points,
	Centroid* centroids, const int num_of_centroids,
	Point* dev_points, Centroid* dev_centroids) {

	static hipError_t cudaStatus;

	cudaStatus = hipMemcpy(dev_points, points, num_of_points * sizeof(Point), hipMemcpyHostToDevice);
	check_cuda_status(&cudaStatus, "hipMemcpy failed! :: classify_points_by_norm_memCopy", __LINE__);

	cudaStatus = hipMemcpy(dev_centroids, centroids, num_of_centroids * sizeof(Centroid), hipMemcpyHostToDevice);
	check_cuda_status(&cudaStatus, "hipMemcpy failed! :: classify_points_by_norm_memCopy", __LINE__);
}

void classify_points_by_norm_free(Point *dev_points, Centroid *dev_centroids)
{
	hipFree(dev_points);
	hipFree(dev_centroids);
}

void classify_points_by_norm(
	Point* points, int num_of_points,
	Centroid* const centroids, const int num_of_centroids,
	const int vector_size) {

	static hipError_t cudaStatus;
	static int max_num_of_thr_per_block = get_max_num_of_threads_per_block();
	static int num_of_blocks = num_of_points / max_num_of_thr_per_block;

	Point *dev_points = NULL;
	Centroid *dev_centroids = NULL;

	classify_points_by_norm_alloc(
		&dev_points, num_of_points,
		&dev_centroids, num_of_centroids);

	classify_points_by_norm_memCopy(
		points, num_of_points,
		centroids, num_of_centroids,
		dev_points, dev_centroids);

	//kernel launch
	calc_norms << <num_of_blocks, max_num_of_thr_per_block >> >(dev_points, dev_centroids, num_of_centroids, vector_size);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	check_cuda_status(&cudaStatus, "hipGetLastError failed! :: classify_points_by_norm", __LINE__);

	// hipDeviceSynchronize waits for the kernel to finish, and returns any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	check_cuda_status(&cudaStatus, "hipDeviceSynchronize failed! :: classify_points_by_norm", __LINE__);

	cudaStatus = hipMemcpy(points, dev_points, num_of_points * sizeof(Point), hipMemcpyDeviceToHost);
	check_cuda_status(&cudaStatus, "hipMemcpy failed! :: classify_points_by_norm", __LINE__);

	classify_points_by_norm_free(dev_points, dev_centroids);
}
////////////////////////////////















