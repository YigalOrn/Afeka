#include "hip/hip_runtime.h"
#include "CUDA_GateWay.h"


__global__ 
	void resetHistoMat(int* dev_histoMat, int bucketsSize)
{
	//reset each thr histo space before use
	for(int i=0;i<bucketsSize;i++)
	{
		dev_histoMat[threadIdx.x*bucketsSize+i] = 0;
	}
}

__global__ 
	void calcHisto(int* dev_dataArr, int* dev_histoMat, int blockSize, int bucketsSize)
{
	//each thread makes his own histo using his chunk of data
	for(int i=0; i<blockSize;i++)
	{
		dev_histoMat[ (threadIdx.x*bucketsSize) + (dev_dataArr[threadIdx.x*blockSize+i]) ] ++; 
	}
}

__global__ 
	void gatherToOne(int* dev_ResultHisto,int* dev_histoMat, int num_thr, int bucketsSize)
{
	for(int i=0;i<num_thr;i++)
	{
		dev_ResultHisto[threadIdx.x] +=dev_histoMat[threadIdx.x+i*bucketsSize];
	}
}

void createLocalHistoMatCuda(int* dev_dataArr, int* dev_localHistoMat, int block_size, int left_units, int num_thr, int bucketsSize, int dataArrSize)
{
	resetHistoMat<<<1, num_thr>>>(dev_localHistoMat, bucketsSize);

	//sync

	calcHisto<<<1, num_thr>>>(dev_dataArr, dev_localHistoMat, block_size, bucketsSize);

	if(left_units != 0)
	{

		//sync

		num_thr = left_units;
		block_size = 1;
		int startIndex = dataArrSize-left_units;
		calcHisto<<<1, num_thr>>>(&dev_dataArr[startIndex], dev_localHistoMat, block_size, bucketsSize);
	}
}

void mergeToFinalHistoArrCuda(int* dev_histoResultArr, int* dev_localHistoMat, int* histoResultArr, int num_thr, int histoResultArrSize)
{
	gatherToOne<<<1, histoResultArrSize>>>(dev_histoResultArr, dev_localHistoMat, num_thr, histoResultArrSize);
}

